#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

using namespace std;

#define ELEMENTS_PER_BUY_OPTION 2
#define STORE_ID_OFFSET 0
#define PRICE_OFFSET 1

#define NUM_PRODUCTS 3
#define NUM_BUY_OPTIONS 6144

#define NUM_THREADS 512 // El número mínimo de threads es 32 (por el tamaño de warp) y el maximo 1024

void printAllProductsAllBuyOptions(int *all_products_buy_options);
void getBestBuyOptions(int *all_products_buy_options, int *best_buy_options);
void printBestBuyOptions(int *best_buy_options);

void initAllProductsBuyOptions(int *all_products_buy_options);
bool areResultsValid(int *all_products_buy_options, int *best_buy_options);

// ToDo: Cada thread ejecuta el kernel.
// Identificar el thread en el que estamos, en base a esto, calculamos si nos toca trabajar y, en caso afirmativo
// comparar entre 2 buy options cuál es la mejor y dejarla en el vector de memoria compartida (temporal)
// Hacemos __syncthreads() para que no haya colisiones y que todos hayan acabado esta ronda de comparación. Pasar a iteración siguiente.
// Cuando se haya agotado el bloque, pasar las opciones del vector temporal al vector de salida en caso de ser el thread 0.
__global__ void KernelKnapsack(unsigned int *total_buy_options, unsigned int *best_buy_options, unsigned int total_buy_options_size)
{
    __shared__ int tmp_best_buy_options[total_buy_options_size];
    unsigned int stride;

    // Cada thread carga 1 elemento desde la memoria global
    unsigned int thread_id = threadIdx.x;
    unsigned int thread_product = blockIdx.x * blockDim.x + threadIdx.x;
    tmp_best_buy_options[thread_id] = total_buy_options[thread_product];
    __syncthreads();

    // Hacemos la reduccion en la memoria compartida
    for(stride = 1; stride < blockDim.x; stride *= 4) {
      if (thread_id % (4 * stride) == 0) {
          // ToDo: Revisar comparación en función de la nueva definición de la estructura del array.
          if (tmp_best_buy_options[thread_id + stride][1] < tmp_best_buy_options[thread_id][1]) {
              tmp_best_buy_options[thread_id] += tmp_best_buy_options[thread_id + stride];
          }
      }
      __syncthreads();
    }

    // El thread 0 escribe el resultado de este bloque en la memoria global
    if (thread_id == 0) {
        best_buy_options[blockIdx.x] = tmp_best_buy_options[0];
    }
}

int main(int argc, char** argv)
{
    // Buy options in host and device
    int *host_all_products_buy_options = (int *) malloc( NUM_PRODUCTS * NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION * sizeof(int) );
    int *best_buy_options = (int *) malloc( NUM_PRODUCTS * ELEMENTS_PER_BUY_OPTION * sizeof(int) );
    unsigned int *device_all_products_buy_options;
    unsigned int *device_best_buy_options;

    // Metadata
    unsigned int buy_option_size = ELEMENTS_PER_BUY_OPTION * sizeof(int);
    unsigned int num_total_buy_options = NUM_PRODUCTS * NUM_BUY_OPTIONS;
    unsigned int total_buy_options_size = num_total_buy_options * buy_option_size;
    unsigned int best_buy_options_size = NUM_PRODUCTS * buy_option_size;

    // Benchmarking
    float elapsed_time;
    hipEvent_t start;
    hipEvent_t stop;

    initAllProductsBuyOptions(host_all_products_buy_options);

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Obtener Memoria en el device
    hipMalloc( (float**) &device_all_products_buy_options, total_buy_options_size );
    hipMalloc( (float**) &device_best_buy_options, best_buy_options_size );

    // Copiar datos desde el host en el device
    hipMemcpy(device_all_products_buy_options, host_all_products_buy_options, total_buy_options_size, hipMemcpyHostToDevice);

    hipEventRecord(start, 0);

    // Ejecutar el kernel (número de bloques = número de productos)
    KernelKnapsack<<<NUM_PRODUCTS, NUM_THREADS>>>(device_all_products_buy_options, device_best_buy_options, total_buy_options_size);

    // Obtener el resultado parcial desde el host
    hipMemcpy(best_buy_options, device_best_buy_options, best_buy_options_size, hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Liberar Memoria del device
    hipFree(device_all_products_buy_options);
    hipFree(device_best_buy_options);

    hipEventElapsedTime(&elapsed_time, start, stop);

    printf("\nKERNEL KNAPSACK\n");
    printf("Vector Size: %d\n", num_total_buy_options);
    printf("Number of Threads: %d\n", NUM_THREADS);
    printf("Number of blocks (products): %d\n", NUM_PRODUCTS);
    printf("Total time %4.6f milseg\n", elapsed_time);
    printf("Bandwidth %4.3f GB/s\n", (num_total_buy_options * sizeof(int)) / (1000000 * elapsed_time));

    hipEventDestroy(start);
    hipEventDestroy(stop);

    // DEBUG
    printAllProductsAllBuyOptions(all_products_buy_options);
    getBestBuyOptions(all_products_buy_options, best_buy_options);
    printBestBuyOptions(best_buy_options);
    // END DEBUG

    if ( areResultsValid( host_all_products_buy_options, best_buy_options ) )
    {
        printf ("TEST PASS\n");
    }
    else
    {
        printf ("TEST FAIL\n");
    }
}

void initAllProductsBuyOptions(int *all_products_buy_options)
{
    // START Vector all the buy options for the product 1

    // product_1_store_1_buy_option;
    all_products_buy_options[0] = 11; // store_id
    all_products_buy_options[1] = 11;  // price

    // product_1_store_2_buy_option;
    all_products_buy_options[2] = 12; // store_id
    all_products_buy_options[3] = 12;  // price

    // END Vector all the buy options for the product 1

    // START Vector all the buy options for the product 2

    // product_1_store_1_buy_option;
    all_products_buy_options[4] = 21; // store_id
    all_products_buy_options[5] = 21;  // price

    // product_1_store_2_buy_option;
    all_products_buy_options[6] = 22; // store_id
    all_products_buy_options[7] = 2;  // price

    // END Vector all the buy options for the product 2
}

bool areResultsValid(int *all_products_buy_options, int *best_buy_options)
{
   int *tmp_best_buy_options;

   getBestBuyOptions(all_products_buy_options, tmp_best_buy_options);

   for (int product_iteration = 0; product_iteration < NUM_PRODUCTS * ELEMENTS_PER_BUY_OPTION; product_iteration += ELEMENTS_PER_BUY_OPTION)
   {
       int current_product_position = product_iteration * NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION;
       int current_product_store_position = current_product_position + STORE_ID_OFFSET;
       int current_product_price_position = current_product_position + PRICE_OFFSET;

       int best_store = best_buy_options[current_product_store_position];
       int best_price = best_buy_options[current_product_price_position];

       int tmp_best_store = tmp_best_buy_options[current_product_store_position];
       int tmp_best_price = tmp_best_buy_options[current_product_price_position];

       if (best_store != tmp_best_store || best_price != tmp_best_price)
       {
           return false;
       }
   }

   return true;
}

void printAllProductsAllBuyOptions(int *all_products_buy_options)
{
    cout << "All products buy options:" << endl;
    for (int i = 0; i < NUM_PRODUCTS; ++i){
        cout << endl << "\tproduct_id: " << i << endl;
        for (int j = 0; j < NUM_BUY_OPTIONS*2; j += 2){
            cout << "Buy option:" << endl;
            cout << "\tstore_id: " << all_products_buy_options[i*NUM_BUY_OPTIONS*2+j] << endl;
            cout << "\tprice: " << all_products_buy_options[i*NUM_BUY_OPTIONS*2+j+1] << endl;
        }
    }
}

void getBestBuyOptions(int *all_products_buy_options, int *best_buy_options)
{
    for(int product_iteration = 0; product_iteration < NUM_PRODUCTS; ++product_iteration)
    {
        int current_product_position = product_iteration * NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION;
        int current_product_store_position = current_product_position + STORE_ID_OFFSET;
        int current_product_price_position = current_product_position + PRICE_OFFSET;

        int best_store = all_products_buy_options[current_product_store_position];
        int best_price = all_products_buy_options[current_product_price_position];

        for(int product_to_compare = ELEMENTS_PER_BUY_OPTION; product_to_compare < NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION; product_to_compare += ELEMENTS_PER_BUY_OPTION)
        {
            if (all_products_buy_options[current_product_position + product_to_compare + PRICE_OFFSET] < best_price)
            {
               best_store = all_products_buy_options[product_iteration * NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION + product_to_compare + STORE_ID_OFFSET];
               best_price = all_products_buy_options[product_iteration * NUM_BUY_OPTIONS * ELEMENTS_PER_BUY_OPTION + product_to_compare + PRICE_OFFSET];
            }
        }
        best_buy_options[product_iteration * ELEMENTS_PER_BUY_OPTION + STORE_ID_OFFSET] = best_store;
        best_buy_options[product_iteration * ELEMENTS_PER_BUY_OPTION + PRICE_OFFSET] = best_price;
    }
}

void printBestBuyOptions(int *best_buy_options)
{
    cout << endl <<"Best products buy options:" << endl;
    for (int i = 0; i < NUM_PRODUCTS*2; i+=2)
    {
        cout << endl << "\tproduct_id: " << i/2 << endl;
        cout << "Buy option:" << endl;
        cout << "\tstore_id: " << best_buy_options[i] << endl;
        cout << "\tprice: " << best_buy_options[i+1] << endl;
    }
}

